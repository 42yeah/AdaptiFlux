#include "hip/hip_runtime.h"
#include "renderstates/streamtube.cuh"
#include <fstream>
#include <random>
#include <imgui.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>
#include "app.cuh"

struct StreamLineVertex
{
    glm::vec3 position;
    glm::vec3 color;
};

struct StreamTubeVertex
{
    glm::vec3 position;
    glm::vec3 normal;
    glm::vec3 color;
};


StreamTubeRenderState::StreamTubeRenderState() : StreamLineRenderState()
{
    streamtube_radius = 1.0f;
    shadow_mapping = false;
    streamtube_graphics_resource = nullptr;
}

StreamTubeRenderState::~StreamTubeRenderState()
{
    StreamLineRenderState::~StreamLineRenderState();
}

void StreamTubeRenderState::initialize(App &app)
{
    StreamLineRenderState::initialize(app);

    if (!allocate_graphics_resources())
    {
        std::cerr << "Failed to allocate streamtube graphics resources?" << std::endl;        
    }

    generate_streamtubes();
}

bool StreamTubeRenderState::allocate_graphics_resources()
{
    int num_streamline_vertices = num_seeds * num_lines * 2;
    // 18 vertices per 2 control points
    int num_streamtube_vertices = (num_streamline_vertices) / 2 * 9;
    // (vertex, normal, color)
    int num_floats = num_streamtube_vertices * 9;
    int size_in_bytes = sizeof(float) * num_floats;

    std::cout << "Allocating " << size_in_bytes << " bytes (" << (size_in_bytes / 1024) << "K) for streamtube rendering." << std::endl;

    std::unique_ptr<float[]> empty_data = std::make_unique<float[]>(num_floats);
    std::memset(empty_data.get(), 0, size_in_bytes);
    streamtube_vao = std::make_shared<VAO>(empty_data.get(), size_in_bytes, GL_DYNAMIC_DRAW, 
        std::vector<VertexAttribPointer>(
            {
                VertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(float) * 9, nullptr),
                VertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, sizeof(float) * 9, (void *) (sizeof(float) * 3)),
                VertexAttribPointer(2, 3, GL_FLOAT, GL_FALSE, sizeof(float) * 9, (void *) (sizeof(float) * 6))
            }
        ),
        GLDrawCall(GL_TRIANGLES, 0, num_streamtube_vertices));

    std::cout << "# streamtube vertices: " << num_streamtube_vertices << std::endl;

    CHECK_CUDA_ERROR(hipGraphicsGLRegisterBuffer(&streamtube_graphics_resource, streamtube_vao->vbo, cudaGraphicsMapFlagsNone));
    
    test_streamtube_generation();

    streamtube_program = Program::make_program("shaders/streamtube.vert", "shaders/streamtube.frag");
    if (streamtube_program == nullptr || !streamtube_program->valid)
    {
        std::cerr << "Invalid streamtube program?" << std::endl;
        return false;
    }
    return true;
}

struct ExportFace
{
    int a, b, c;
};

bool StreamTubeRenderState::export_streamtube_vbo_as_obj(const std::string &path)
{
    glBindVertexArray(streamtube_vao->vao);
    glBindBuffer(GL_ARRAY_BUFFER, streamtube_vao->vbo);
    StreamTubeVertex *verts = (StreamTubeVertex *) (glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY));
    int num_verts = streamtube_vao->draw_call.size;

    std::ofstream output(path);
    if (!output.good())
    {
        std::cerr << "Bad output: " << path << "?" << std::endl;
    }

    // Since there are 200 num_seeds, there should be 200 tubes
    int current_vertex_id = 1;
    std::vector<StreamTubeVertex> vertices;
    std::vector<ExportFace> faces;
    for (int i = 0; i < num_seeds; i++)
    {
        int base_vertices_offset = i * (num_lines * 9);
        
        int j = 0;
        while (true)
        {
            StreamTubeVertex v1 = verts[base_vertices_offset + j + 0];
            StreamTubeVertex v2 = verts[base_vertices_offset + j + 1];
            StreamTubeVertex v3 = verts[base_vertices_offset + j + 2];
            int num_zero_verts = 0.0f;
            num_zero_verts += v1.position == glm::vec3(0.0f) ? 1 : 0;
            num_zero_verts += v2.position == glm::vec3(0.0f) ? 1 : 0;
            num_zero_verts += v3.position == glm::vec3(0.0f) ? 1 : 0;
            assert(num_zero_verts == 3 || num_zero_verts == 0); // An incomplete vertex means this assert will fail
            if (num_zero_verts == 3)
            {
                break;
            }
            vertices.insert(vertices.end(), {v1, v2, v3});
            faces.insert(faces.end(), {current_vertex_id, current_vertex_id + 1, current_vertex_id + 2});
            current_vertex_id += 3;

            output << "v " << v1.position.x << " " << v1.position.y << " " << v1.position.z << std::endl;
            output << "v " << v2.position.x << " " << v2.position.y << " " << v2.position.z << std::endl;
            output << "v " << v3.position.x << " " << v3.position.y << " " << v3.position.z << std::endl;
            j += 3;
        }
    }

    for (int i = 0; i < vertices.size(); i += 3)
    {
        output << "vt 0.0 0.0" << std::endl;
        output << "vt 0.0 0.0" << std::endl;
        output << "vt 0.0 0.0" << std::endl;
    }

    for (int i = 0; i < vertices.size(); i += 3)
    {
        output << "vn " << vertices[i].normal.x << " " << vertices[i].normal.y << " " << vertices[i].normal.z << std::endl;
        output << "vn " << vertices[i + 1].normal.x << " " << vertices[i + 1].normal.y << " " << vertices[i + 1].normal.z << std::endl;
        output << "vn " << vertices[i + 2].normal.x << " " << vertices[i + 2].normal.y << " " << vertices[i + 2].normal.z << std::endl;
    }

    for (const ExportFace &f : faces)
    {
        output << "f " << f.a << "/" << f.a << "/" << f.a << " " << f.b << "/" << f.b << "/" << f.b << " " << f.c << "/" << f.c << "/" << f.c << std::endl;
    }
    output.close();

    glUnmapBuffer(GL_ARRAY_BUFFER);
    return true;
}

void StreamTubeRenderState::destroy()
{
    StreamLineRenderState::destroy();

    CHECK_CUDA_ERROR(hipGraphicsUnregisterResource(streamtube_graphics_resource));
    streamtube_graphics_resource = nullptr;
}

void StreamTubeRenderState::render(App &app)
{
    streamtube_program->use();
    glUniformMatrix4fv(streamtube_program->at("model"), 1, GL_FALSE, glm::value_ptr(glm::mat4(1.0f)));
    glUniformMatrix4fv(streamtube_program->at("view"), 1, GL_FALSE, glm::value_ptr(app.camera.view));
    glUniformMatrix4fv(streamtube_program->at("perspective"), 1, GL_FALSE, glm::value_ptr(app.camera.perspective));
    streamtube_vao->draw();

    if (render_seed_points)
    {
        glPointSize(point_size);
        seed_points_program->use();
        glUniformMatrix4fv(seed_points_program->at("model"), 1, GL_FALSE, glm::value_ptr(glm::mat4(1.0f)));
        glUniformMatrix4fv(seed_points_program->at("view"), 1, GL_FALSE, glm::value_ptr(app.camera.view));
        glUniformMatrix4fv(seed_points_program->at("perspective"), 1, GL_FALSE, glm::value_ptr(app.camera.perspective));
        seed_points_vao->draw();
    }
}

void StreamTubeRenderState::process_events(App &app)
{
    StreamLineRenderState::process_events(app);
}

void StreamTubeRenderState::key_pressed(App &app, int key)
{
    StreamLineRenderState::key_pressed(app, key);
    switch (key)
    {
        case GLFW_KEY_R:
        case GLFW_KEY_T:
        case GLFW_KEY_O:
        case GLFW_KEY_P:
        case GLFW_KEY_LEFT_BRACKET:
        case GLFW_KEY_RIGHT_BRACKET:
        case GLFW_KEY_PERIOD:
            generate_streamtubes();
            break;
    }
}

void StreamTubeRenderState::draw_user_controls(App &app)
{    
    ImGui::SetNextWindowPos({220.0f, 0.0f}, ImGuiCond_FirstUseEver);
    ImGui::SetNextWindowSize({app.screen_width - 220.0f, 140}, ImGuiCond_FirstUseEver);
    
    bool should_update = false;

    if (ImGui::Begin("Streamtube Controls"))
    {
        should_update |= ImGui::SliderFloat("Simulation delta time", &simulation_dt, 0.001f, 1.0f);
        if (ImGui::Button("Reset"))
        {
            simulation_dt = 1.0f / 256.0f;
            should_update = true;
        }
        if (ImGui::RadioButton("Delta wing recommended strategy", seed_points_strategy == 0)) { seed_points_strategy = 0; should_update = true; }
        if (ImGui::RadioButton("Line", seed_points_strategy == 1)) { seed_points_strategy = 1; should_update = true; }
        if (ImGui::RadioButton("Rect", seed_points_strategy == 2)) { seed_points_strategy = 2; should_update = true; }
        if (seed_points_strategy != 0 && ImGui::CollapsingHeader("Seeding strategy"))
        {
            ImGui::Text("Bounding box: (%f %f %f)", app.delta_wing_bounding_box.max.x,
                app.delta_wing_bounding_box.max.y,
                app.delta_wing_bounding_box.max.z);

            should_update |= ImGui::InputFloat3("Seed begin", (float *) &seed_begin);
            should_update |= ImGui::InputFloat3("Seed end", (float *) &seed_end);
            ImGui::Text("Seeding plane offset axis");
        }
        if (seed_points_strategy == 0)
        {
            should_update |= ImGui::SliderFloat("Seeding plane (X axis)", &seeding_plane_x, 0.0f, app.res.vf_tex.extent.width);
            if (ImGui::Button("Go to critical region"))
            {
                seeding_plane_x = 51.0f;
                should_update = true;
            }
        }
        should_update |= ImGui::Checkbox("Use Runge-Kutta 4 integrator", &use_runge_kutta_4_integrator);
        should_update |= ImGui::Checkbox("Adaptive seeding", &adaptive_mode);

        if (adaptive_mode)
        {
            if (ImGui::CollapsingHeader("Adaptive mode properties"))
            {
                // should_update |= ImGui::SliderFloat("Seed point generation threshold", &seed_point_threshold, 0.001f, app.res.vf_tex.longest_vector);
                should_update |= ImGui::SliderFloat("Seed point generation threshold", &seed_point_threshold, 0.001f, 1.0f);
                should_update |= ImGui::SliderFloat("Adaptive explosion radius", &adaptive_explosion_radius, 1.0f, 20.0f);
                should_update |= ImGui::SliderInt("Number of explosions", &num_explosion, 1, 10);
                should_update |= ImGui::SliderInt("Explosion cooldown counter", &explosion_cooldown_counter, 1, 4000);
            }
        }

        should_update |= ImGui::Checkbox("Streamtube simplification", &do_simplify);
        if (do_simplify)
        {
            if (ImGui::CollapsingHeader("Simplification properties"))
            {
                should_update |= ImGui::SliderFloat("Simplification threshold", &distortion_threshold, 1.001f, 1.5f);
            }
        }

        ImGui::Checkbox("Render seed points", &render_seed_points);
        if (render_seed_points)
        {
            ImGui::SliderFloat("Seed point point size", &point_size, 1.0f, 20.0f);
        }
        
        should_update |= ImGui::SliderFloat("Streamtube radius", &streamtube_radius, 0.1f, 10.0f);
        should_update |= ImGui::Checkbox("Enable shadow mapping", &shadow_mapping);

        ImGui::End();

        if (should_update)
        {
            generate_streamlines(app);
            generate_streamtubes();
        }
    }
}


//
// Stores information about the creation of streamtube.
//
struct StreamTubeInfo
{
    int streamline_starting_index;
    int streamtube_starting_index;
    int streamline_index;
    int streamtube_index;
};

__global__ void streamtube_kernel(float *streamtube_vbo_data,
                                  size_t streamtube_stride,
                                  float *streamline_vbo_data,
                                  size_t streamline_stride,
                                  int num_seeds,
                                  float streamtube_radius,
                                  StreamTubeInfo *info)
{
    int seed_index = blockIdx.y * gridDim.x + blockIdx.x;
    if (seed_index >= num_seeds)
    {
        return;
    }

    int streamline_starting_index = seed_index * streamline_stride;
    int streamtube_starting_index = seed_index * streamtube_stride;
    int streamline_index = streamline_starting_index;
    int streamtube_index = streamtube_starting_index;

    memset(&streamtube_vbo_data[streamtube_starting_index], 0, sizeof(float) * streamtube_stride);

    glm::vec3 up = glm::vec3(0.0f, 1.0f, 0.0f);
    unsigned int order[18] = 
    {
        0, 1, 4,
        0, 4, 3,
        0, 2, 5,
        0, 5, 3,
        1, 2, 4,
        4, 2, 5
    };
    while (true)
    {
        StreamLineVertex &streamline_vert_a = (*(StreamLineVertex *) &(streamline_vbo_data[streamline_index]));
        StreamLineVertex &streamline_vert_b = (*(StreamLineVertex *) &(streamline_vbo_data[streamline_index + 6]));

        if (streamline_vert_a.position == glm::vec3(0.0f) || streamline_vert_b.position == glm::vec3(0.0f))
        {
            break;
        }

        glm::vec3 front = glm::normalize(streamline_vert_b.position - streamline_vert_a.position);
        // TODO: cross operation might take a long time.
        glm::vec3 right = glm::normalize(glm::cross(front, up));
        glm::vec3 up = glm::normalize(glm::cross(right, front));

        // Three (3) new vertices spawns from each. A empty triangular prism will be formed.
        StreamTubeVertex tube_vertices[6];
        for (int i = 0; i < 3; i++)
        {
            float rot = ((float) (i + 1) / 3) * 2.0f * glm::pi<float>();
            glm::vec3 tube_left_pos = streamline_vert_a.position + streamtube_radius * (right * cosf(rot) + up * sinf(rot));
            glm::vec3 tube_right_pos = streamline_vert_b.position + streamtube_radius * (right * cosf(rot) + up * sinf(rot));
            StreamTubeVertex left;
            left.position = tube_left_pos;
            left.normal = glm::vec3(0.0f); // TODO: TBD
            left.color = streamline_vert_a.color;
            StreamTubeVertex right;
            right.position = tube_right_pos;
            right.normal = glm::vec3(0.0f);
            right.color = streamline_vert_b.color;
            tube_vertices[i] = left;
            tube_vertices[i + 3] = right;
        }
        
        StreamTubeVertex *indices = (StreamTubeVertex *) &(streamtube_vbo_data[streamtube_index]);
        for (int i = 0; i < 18; i++)
        {
            indices[i] = tube_vertices[order[i]];
        }

        streamline_index += 2 * 6;
        streamtube_index += 18 * 9;
    }

    info[seed_index].streamline_starting_index = streamline_starting_index;
    info[seed_index].streamline_index = streamline_index;
    info[seed_index].streamtube_starting_index = streamtube_starting_index;
    info[seed_index].streamtube_index = streamtube_index;

}

bool StreamTubeRenderState::generate_streamtubes()
{
    int num_blocks_x = 32;
    int num_blocks_y = (num_seeds + num_blocks_x - 1) / num_blocks_x;
    
    std::cout << "Streamtube generation report: block count: " << num_blocks_x << "x" << num_blocks_y << std::endl;
    dim3 num_blocks(num_blocks_x, num_blocks_y, 1);

    // Map streamline and streamtube data.
    float *streamline_vbo_data;
    size_t mapped_size;
    size_t streamline_stride = num_lines * 2 * 6;
    size_t expected = num_seeds * streamline_stride * sizeof(float);
    CHECK_CUDA_ERROR(hipGraphicsMapResources(1, &streamline_graphics_resource));
    CHECK_CUDA_ERROR(hipGraphicsResourceGetMappedPointer((void **) (&streamline_vbo_data), &mapped_size, streamline_graphics_resource));
    std::cout << "Mapped size: " << mapped_size << " bytes as opposed to the expected of " << expected << std::endl;

    float *streamtube_vbo_data;
    size_t streamtube_stride = num_lines * 9 * 9;
    CHECK_CUDA_ERROR(hipGraphicsMapResources(1, &streamtube_graphics_resource));
    CHECK_CUDA_ERROR(hipGraphicsResourceGetMappedPointer((void **) (&streamtube_vbo_data), &mapped_size, streamtube_graphics_resource));
    expected = num_seeds * streamtube_stride * sizeof(float);
    std::cout << "Mapped size: " << mapped_size << " bytes as opposed to the expected of " << expected << std::endl;

    std::unique_ptr<StreamTubeInfo[]> info = std::make_unique<StreamTubeInfo[]>(num_blocks.x * num_blocks.y);
    StreamTubeInfo *info_cuda = nullptr;
    CHECK_CUDA_ERROR(hipMalloc(&info_cuda, num_blocks.x * num_blocks.y * sizeof(StreamTubeInfo)));
    CHECK_CUDA_ERROR(hipMemcpy(info_cuda, info.get(), num_blocks.x * num_blocks.y * sizeof(StreamTubeInfo), hipMemcpyHostToDevice));

    streamtube_kernel<<<num_blocks, 1>>>(streamtube_vbo_data, streamtube_stride, streamline_vbo_data, streamline_stride, num_seeds, streamtube_radius, info_cuda);

    CHECK_CUDA_ERROR(hipMemcpy(info.get(), info_cuda, num_blocks.x * num_blocks.y * sizeof(StreamTubeInfo), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipFree(info_cuda));

    // for (int i = 0; i < num_blocks.y; i++)
    // {
    //     for (int j = 0; j < num_blocks.x; j++)
    //     {
    //         int idx = i * num_blocks.x + j;

    //         std::cout << "BLK " << idx << ": SLSI " << info[idx].streamline_starting_index << " (" << (info[idx].streamline_starting_index / 6) << "); " << 
    //             "SLI " << info[idx].streamline_index << " (" << (info[idx].streamline_index / 6) << "); " <<
    //             "STSI " << info[idx].streamtube_starting_index << " (" << (info[idx].streamtube_starting_index / 9) << "); " <<
    //             "STI " << info[idx].streamtube_index << " (" << (info[idx].streamtube_index / 9) << ")" << std::endl; 
    //     }
    // }

    CHECK_CUDA_ERROR(hipGraphicsUnmapResources(1, &streamline_graphics_resource));
    CHECK_CUDA_ERROR(hipGraphicsUnmapResources(1, &streamtube_graphics_resource));
    return true;
}

bool StreamTubeRenderState::generate_bare_bones_streamline()
{
    glBindVertexArray(streamline_vao->vao);
    glBindBuffer(GL_ARRAY_BUFFER, streamline_vao->vbo);
    StreamLineVertex *verts = (StreamLineVertex *) (glMapBuffer(GL_ARRAY_BUFFER, GL_WRITE_ONLY));
    size_t verts_size_in_bytes = sizeof(StreamLineVertex) * num_seeds * num_lines * 2;
    std::memset(verts, 0, verts_size_in_bytes);
    std::cout << "Clearing up " << verts_size_in_bytes << " bytes (" << (verts_size_in_bytes / sizeof(float)) << " floats)" << std::endl;

    StreamLineVertex replacements[6];
    
    std::uniform_real_distribution<float> distrib;
    std::random_device dev;

    for (int i = 0; i < 6; i++)
    {
        replacements[i].position = glm::vec3(distrib(dev) * 100.0f, distrib(dev) * 100.0f, distrib(dev) * 100.0f);
        replacements[i].color = glm::vec3(distrib(dev), distrib(dev), distrib(dev));
    }
    for (int i = 0; i < 5; i++)
    {
        verts[i * 2 + 0] = replacements[i];
        verts[i * 2 + 1] = replacements[i + 1];
    }

    glUnmapBuffer(GL_ARRAY_BUFFER);
    return true;
}

bool StreamTubeRenderState::test_streamtube_generation()
{
    if (!generate_bare_bones_streamline())
    {
        std::cerr << "Cannot generate bare bones streamline." << std::endl;
        return false;
    }
    if (!generate_streamtubes())
    {
        std::cerr << "Cannot generate streamtubes." << std::endl;
        return false;
    }
    if (!export_streamtube_vbo_as_obj("test.obj"))
    {
        std::cerr << "Failed to export streamtube obj." << std::endl;
        return false;
    }
    return true;
}
