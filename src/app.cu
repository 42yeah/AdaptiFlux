#include "hip/hip_runtime.h"
#include "app.cuh"
#include <cfloat>
#include <glm/gtc/type_ptr.hpp>
#include <fstream>
#include <iomanip>
#include <filesystem>
#include <string>
#include <imgui.h>
#include <imgui_impl_glfw.h>
#include <imgui_impl_opengl3.h>
#include <implot.h>
#include "debug_kernels.cuh"
#include "renderstates/lineglyph.cuh"
#include "renderstates/arrowglyph.cuh"
#include "renderstates/streamline.cuh"
#include "renderstates/streamtube.cuh"

// For GLFW stuffs
App *bound_app = nullptr;

App::App(YYLVVRes &res) : res(res),
    window(res.window),
    valid(false),
    user_interface_mode(false),
    visualization_mode(0),
    should_draw_bounding_box(true),
    should_draw_delta_wing(true),
    should_draw_shadow(true),
    custom_resolution(false)
{
    if (!init())
    {
        std::cerr << "Failed to initialize UI & its resources?" << std::endl;
        return;
    }
    valid = true;
}

App::~App()
{
    if (render_state) 
    {
        render_state->destroy();
    }

    std::cout << "Destroying the rest of the app..." << std::endl;
    
    CHECK_CUDA_ERROR(hipDestroyTextureObject(ctf_tex_cuda));
    CHECK_CUDA_ERROR(hipFreeArray(ctf_data_cuda));

    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImPlot::DestroyContext();
    ImGui::DestroyContext();
}

bool App::init()
{
    std::cout << "Initializing bounding box and bounding box program." << std::endl;
    const glm::vec3 a = glm::vec3(-0.5f, -0.5f, -0.5f),
                b = glm::vec3(0.5f, -0.5f, -0.5f),
                c = glm::vec3(0.5f, -0.5f, 0.5f),
                d = glm::vec3(-0.5f, -0.5f, 0.5f),
                e = a + glm::vec3(0.0f, 1.0f, 0.0f),
                f = b + glm::vec3(0.0f, 1.0f, 0.0f),
                g = c + glm::vec3(0.0f, 1.0f, 0.0f),
                h = d + glm::vec3(0.0f, 1.0f, 0.0f);

    glm::vec3 bounding_box_data[] = 
    {
            a, b, b, c, c, d, d, a,
            e, f, f, g, g, h, h, e,
            a, b, b, f, f, e, e, a,
            d, c, c, g, g, h, h, d,
            a, d, d, h, h, e, e, a,
            b, c, c, g, g, f, f, b
    };

    bounding_box_vao = VAO::make_vao(bounding_box_data,
                                     sizeof(bounding_box_data),
                                     GL_STATIC_DRAW,
                                     {VertexAttribPointer(0,
                                                          3,
                                                          GL_FLOAT,
                                                          GL_FALSE,
                                                          sizeof(float) * 3,
                                                          nullptr)},
                                     GLDrawCall(GL_LINES, 0, 48));

    std::cout << "Compiling line drawing program." << std::endl;
    bounding_box_program = Program::make_program("shaders/lines.vert", "shaders/lines.frag");
    
    if (!bounding_box_program || !bounding_box_program->valid) 
    {
        std::cerr << "Cannot link line-drawing program?" << std::endl;
        return false;
    }

    std::cout << "Initializing shadow floor VAO and the framebuffer." << std::endl;
    glm::vec3 shadow_floor_data[] = {
        a, b, c, c, d, a
    };
    shadow_floor = VAO::make_vao(shadow_floor_data,
                                 sizeof(shadow_floor_data),
                                 GL_STATIC_DRAW,
                                 {
                                     VertexAttribPointer(0,
                                                         3,
                                                         GL_FLOAT,
                                                         GL_FALSE,
                                                         sizeof(float) * 3,
                                                         nullptr)
                                 },
                                 GLDrawCall(GL_TRIANGLES, 0, 6));

    shadow_floor_program = Program::make_program("shaders/shadow_floor.vert",
                                                 "shaders/shadow_floor.frag");

    if (!shadow_floor_program || !shadow_floor_program->valid)
    {
        std::cerr << "Cannot link shadow floor rendering program?" << std::endl;
        return false;
    }

    std::cout << "Configuring OpenGL & GLFW." << std::endl;
    glEnable(GL_DEPTH_TEST);
    glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
    glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
    glfwSetKeyCallback(window, key_callback_glfw);
    glfwSetCursorPosCallback(window, cursor_pos_callback_glfw);
    glfwSetWindowSizeCallback(window, window_size_callback_glfw);
    render_state = nullptr;
    glfwGetFramebufferSize(window, &screen_width, &screen_height);
    glfwSwapInterval(1);
    last_instant = glfwGetTime();
    delta_time = 0.0f;

    // Initialize delta wing
    std::cout << "Evaluating delta wing bounding box and allocating graphics resources." << std::endl;
    delta_wing_bounding_box = res.vf_tex.get_bounding_box();
    float x_min = 50.0f;
    float x_max = 169.0f;
    float y_min = 27.5f;
    float y_mid = 100.0f;
    float y_max = 172.5f;
    float z = 0.0f;
    std::vector<float3> delta_wing_fl3 = 
    {
        make_float3(x_min, y_mid, z),
        make_float3(x_max, y_min, z),
        make_float3(x_max, y_max, z)
    };
    std::cout << "Compiling delta wing shader." << std::endl;
    delta_wing_vao = VAO::make_vao(delta_wing_fl3.data(),
                                   delta_wing_fl3.size() * sizeof(float3),
                                   GL_STATIC_DRAW,
                                   {
                                       VertexAttribPointer(0, 3, GL_FLOAT,
                                                           GL_FALSE, sizeof(float) * 3,
                                                           nullptr)
                                   },
                                   GLDrawCall(GL_TRIANGLES, 0, 3));
    delta_wing_program = Program::make_program("shaders/lines.vert", "shaders/delta.frag");
    
    if (!delta_wing_program || !delta_wing_program->valid) 
    {
        return false;
    }

    // Color transfer function (CTF)
    std::vector<float4> vector_magnitude_ctf;
    std::cout << "Initializing color transfer function for CUDA: creating device array." << std::endl;
    vector_magnitude_ctf.push_back(make_float4(0.4f, 0.6f, 0.9f, 1.0f)); // TODO: 1D texture is weird because it has a
    vector_magnitude_ctf.push_back(make_float4(0.0f, 1.0f, 0.0f, 1.0f)); //       weird padding of 0.5
    vector_magnitude_ctf.push_back(make_float4(0.9f, 0.9f, 0.0f, 1.0f));
    vector_magnitude_ctf.push_back(make_float4(1.0f, 0.0f, 0.0f, 1.0f));
    int vm_size_in_bytes = vector_magnitude_ctf.size() * sizeof(float4);
    hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
    CHECK_CUDA_ERROR(hipMallocArray(&ctf_data_cuda, &desc, vector_magnitude_ctf.size(), 0, 0));
    CHECK_CUDA_ERROR(hipMemcpy2DToArray(ctf_data_cuda, 0, 0, vector_magnitude_ctf.data(), vm_size_in_bytes, vm_size_in_bytes, 1, hipMemcpyHostToDevice));

    std::cout << "Creating vector magnitude CTF texture." << std::endl;
    hipResourceDesc rdesc;
    std::memset(&rdesc, 0, sizeof(hipResourceDesc));
    {
        rdesc.resType = hipResourceTypeArray;
        rdesc.res.array.array = ctf_data_cuda;
    }
    hipTextureDesc tdesc;
    std::memset(&tdesc, 0, sizeof(hipTextureDesc));
    {
        tdesc.addressMode[0] = hipAddressModeWrap;
        tdesc.filterMode = hipFilterModeLinear;
        tdesc.readMode = hipReadModeElementType;
        tdesc.normalizedCoords = 0; // let's try normalizing it
    }
    CHECK_CUDA_ERROR(hipCreateTextureObject(&ctf_tex_cuda, &rdesc, &tdesc, nullptr));
    std::cout << "Vector magnitude CTF texture creation complete." << std::endl;

    align_camera();

    // Setup ImGui
    ImGui::CreateContext();
    ImPlot::CreateContext();
    ImGuiIO &io = ImGui::GetIO();
    io.IniFilename = nullptr;

    ImGui::StyleColorsLight();

    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init("#version 330 core");

    // Setup framebuffer
    framebuffer = std::make_unique<Framebuffer>(screen_width, screen_height);
    float rect_vao_data[] = {
        -1.0f, -1.0f, 0.0f,
        1.0f, -1.0f, 0.0f,
        1.0f, 1.0f, 0.0f,
        1.0f, 1.0f, 0.0f,
        -1.0f, 1.0f, 0.0f,
        -1.0f, -1.0f, 0.0f
    };
    rect_vao = VAO::make_vao(rect_vao_data, sizeof(rect_vao_data), GL_STATIC_DRAW, 
        std::vector<VertexAttribPointer>(
            {
                VertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(float) * 3, nullptr)
            }
        ), GLDrawCall(GL_TRIANGLES, 0, 6));
    framebuffer_render_program = Program::make_program("shaders/simple.vert", "shaders/framebuffer.frag");
    custom_resolution = false;
    // assert(framebuffer->render_test_buffer());
    CHECK_OPENGL_ERRORS();

    elapsed = 0.0f;
    framerate_sum = 0.0f;
    framerate_history.framerate.clear();
    framerate_history.delta_time.clear();
    framerate_history.timestamp.clear();
    for (int i = 0; i < MAX_FRAMERATE_HISTORY; i++)
    {
        framerate_history.history_xs[i] = ((float) i / MAX_FRAMERATE_HISTORY);
    }
    std::memset(framerate_history.history_ys, 0, sizeof(float) * MAX_FRAMERATE_HISTORY);
    framerate_history.best_framerate = std::numeric_limits<float>::min();
    framerate_history.worst_framerate = std::numeric_limits<float>::max();
    framerate_history.stress_test = false;

    return true;
}

void App::key_callback_glfw(GLFWwindow *window, int key, int scancode, int action, int mods)
{
    assert(bound_app != nullptr);
    bound_app->key_callback(window, key, scancode, action, mods);
}

void App::cursor_pos_callback_glfw(GLFWwindow *window, double xpos, double ypos)
{
    assert(bound_app != nullptr);
    bound_app->cursor_pos_callback(window, xpos, ypos);
}

void App::window_size_callback_glfw(GLFWwindow *window, int width, int height)
{
    assert(bound_app != nullptr);
    bound_app->window_size_callback(window, width, height);
}

void App::key_callback(GLFWwindow *window, int key, int scancode, int action, int mods)
{
    if (action != 1) 
    {
        return;
    }

    if (user_interface_mode && key == GLFW_KEY_SPACE)
    {
        set_user_interface_mode(false);
        return;
    }

    switch (key) 
    {
        case GLFW_KEY_L:
            switch_state(std::make_shared<LineGlyphRenderState>());
            visualization_mode = 1;
            break;

        case GLFW_KEY_G:
            switch_state(std::make_shared<ArrowGlyphRenderState>());
            visualization_mode = 2;
            break;

        case GLFW_KEY_Z:
            switch_state(std::make_shared<StreamLineRenderState>());
            visualization_mode = 3;
            break;

        case GLFW_KEY_SPACE:
            set_user_interface_mode(!user_interface_mode);
            break;

        case GLFW_KEY_GRAVE_ACCENT:
            debug_vf();
            break;
    }

    if (render_state) 
    {
        render_state->key_pressed(*this, key);
    }
}

void App::cursor_pos_callback(GLFWwindow *window, double xpos, double ypos)
{
    ypos = -ypos;
    xpos = -xpos;
    
    if (!camera.prev_cursor_pos || user_interface_mode) 
    {
        camera.prev_cursor_pos = glm::dvec2(xpos, ypos);
        return;
    }

    glm::dvec2 curr_pos = glm::dvec2(xpos, ypos);
    glm::dvec2 delta_pos = curr_pos - *camera.prev_cursor_pos;
    camera.yaw += delta_pos.x * camera.sensitivity;
    camera.pitch += delta_pos.y * camera.sensitivity;
    camera.prev_cursor_pos = curr_pos;
    camera.update_components(screen_width, screen_height);
}

void App::window_size_callback(GLFWwindow *window, int width, int height)
{
    screen_width = width;
    screen_height = height;
    glViewport(0, 0, screen_width, screen_height);
    if (custom_resolution)
    {
        // Nothing ever changes
        return;
    }
    framebuffer->resize(screen_width, screen_height);
}

void App::align_camera()
{
    glm::vec3 extent = delta_wing_bounding_box.extend(); // TODO: a typo
    float max_ext = glm::max(glm::max(extent.x, extent.y), extent.z);
    float init_dist = glm::max(extent.x, extent.y) * 0.5f * sqrt(3.0f);
    camera.eye = delta_wing_bounding_box.center() - glm::vec3(0.0f, 0.0f, init_dist);
    camera.speed = max_ext * 0.1f; // whole thing in 10 seconds
    camera.z_near = 1.0f;
    camera.z_far = max_ext * 2.0f + fabs(init_dist);
    camera.update_components(screen_width, screen_height);
}

void App::handle_continuous_key_events()
{
    if (user_interface_mode)
    {
        return;
    }

    if (glfwGetKey(window, GLFW_KEY_W)) 
    {
        camera.eye += camera.front * camera.speed * delta_time;
    }
    if (glfwGetKey(res.window, GLFW_KEY_S)) 
    {
        camera.eye -= camera.front * camera.speed * delta_time;
    }
    if (glfwGetKey(res.window, GLFW_KEY_A)) 
    {
        camera.eye -= camera.right * camera.speed * delta_time;
    }
    if (glfwGetKey(res.window, GLFW_KEY_D)) 
    {
        camera.eye += camera.right * camera.speed * delta_time;
    }
}

void App::loop()
{
    while (!glfwWindowShouldClose(window)) 
    {
        if (!valid)
        {
            glfwSetWindowShouldClose(window, GLFW_TRUE);
        }

        bound_app = this;
        glfwPollEvents();

        double this_instant = glfwGetTime();
        delta_time = (float) (this_instant - last_instant);
        elapsed += delta_time;
        last_instant = this_instant;

        handle_continuous_key_events();

        if (render_state) 
        {
            render_state->process_events(*this);
        }

        glfwGetFramebufferSize(window, &screen_width, &screen_height);
        glViewport(0, 0, screen_width, screen_height);

        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        framebuffer->use();
        if (custom_resolution)
        {
            camera.update_components(custom_resolution_size.x, custom_resolution_size.y);
        }
        else
        {
            camera.update_components(screen_width, screen_height);
        }
        draw_delta_wing();

        if (render_state) 
        {
            render_state->render(*this);
        }
        framebuffer->done();

        framebuffer->draw(rect_vao, framebuffer_render_program, "fbo", 0);

        if (user_interface_mode)
        {
            ImGui_ImplOpenGL3_NewFrame();
            ImGui_ImplGlfw_NewFrame();
            ImGui::NewFrame();

            draw_user_controls();
            if (render_state)
            {
                render_state->draw_user_controls(*this);
            }

            ImGui::Render();
            ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
        }

        glfwSwapBuffers(window);
    }
}

void App::draw_delta_wing() const
{
    if (should_draw_bounding_box)
    {
        // 1. Draw the bounding box (that we calculated)
        bounding_box_program->use();
        glm::mat4 model = glm::translate(glm::mat4(1.0f), delta_wing_bounding_box.center());
        model = glm::scale(model, delta_wing_bounding_box.extend());
        glUniformMatrix4fv(bounding_box_program->at("model"), 1, GL_FALSE, glm::value_ptr(model));
        glUniformMatrix4fv(bounding_box_program->at("view"), 1, GL_FALSE, glm::value_ptr(camera.view));
        glUniformMatrix4fv(bounding_box_program->at("perspective"), 1, GL_FALSE, glm::value_ptr(camera.perspective));
        bounding_box_vao->draw();
    }

    if (should_draw_delta_wing)
    {
        // 2. Draw the delta wing triangle
        delta_wing_program->use();
        glUniformMatrix4fv(delta_wing_program->at("model"), 1, GL_FALSE, glm::value_ptr(glm::mat4(1.0f)));
        glUniformMatrix4fv(delta_wing_program->at("view"), 1, GL_FALSE, glm::value_ptr(camera.view));
        glUniformMatrix4fv(delta_wing_program->at("perspective"), 1, GL_FALSE, glm::value_ptr(camera.perspective));
        delta_wing_vao->draw();
    }
}

void App::switch_state(std::shared_ptr<RenderState> new_state)
{
    if (render_state != nullptr) 
    {
        render_state->destroy();
    }
    
    render_state = new_state;
    if (new_state)
    {
        render_state->initialize(*this);    
    }
}

void App::draw_user_controls()
{
    static std::vector<std::string> supported = 
    {
        "No visualizations",
        "Line glyphs",
        "Arrow glyphs",
        "Streamlines",
        "Streamtubes"
    };

    ImGui::SetNextWindowPos({0, 0}, ImGuiCond_FirstUseEver);
    ImGui::SetNextWindowSize({220, 200}, ImGuiCond_FirstUseEver);
    if (ImGui::Begin("Visualizations"))
    {
        ImGui::Text("Visualization mode");
        ImGui::SetNextItemWidth(-FLT_MIN);
        if (ImGui::BeginListBox("Streamline Visualizations List"))
        {
            for (int i = 0; i < supported.size(); i++)
            {
                bool is_selected = visualization_mode == i;

                if (ImGui::Selectable(supported[i].c_str(), is_selected))
                {
                    visualization_mode = i;
                    switch (visualization_mode)
                    {
                        case 0:
                            switch_state(nullptr);
                            break;

                        case 1:
                            switch_state(std::make_shared<LineGlyphRenderState>());
                            break;

                        case 2:
                            switch_state(std::make_shared<ArrowGlyphRenderState>());
                            break;

                        case 3:
                            switch_state(std::make_shared<StreamLineRenderState>());
                            break;

                        case 4:
                            switch_state(std::make_shared<StreamTubeRenderState>());
                            break;
                    }
                }

                if (is_selected)
                {
                    ImGui::SetItemDefaultFocus();
                }
            }
            ImGui::EndListBox();
        }
    }
    ImGui::End();

    ImGui::SetNextWindowPos({0, 200}, ImGuiCond_FirstUseEver);
    ImGui::SetNextWindowSize({250, 180}, ImGuiCond_FirstUseEver);
    if (ImGui::Begin("Camera configurations"))
    {
        // bool should_update_camera = false;
        float camera_eye[3] = {camera.eye.x, camera.eye.y, camera.eye.z};
        float camera_py[2] = {camera.pitch, camera.yaw};

        ImGui::Text("Camera position");
        ImGui::SetNextItemWidth(-FLT_MIN);
        ImGui::InputFloat3("##position", camera_eye);
        ImGui::Text("Pitch & yaw");
        ImGui::SetNextItemWidth(-FLT_MIN);
        ImGui::InputFloat2("##pitchyaw", camera_py);
        ImGui::Checkbox("Bounding box", &should_draw_bounding_box);
        ImGui::Checkbox("Delta wing", &should_draw_delta_wing);
        ImGui::Checkbox("Shadow map", &should_draw_shadow);

        if (ImGui::Button("Favorite camera pose"))
        {
            favorite_camera_pose();
        }
        ImGui::SameLine();
        if (ImGui::Button("Restore"))
        {
            restore_camera_pose();
        }

        if (ImGui::Button("Screenshot"))
        {
            screenshot();
        }
        if (ImGui::Checkbox("Custom resolution", &custom_resolution))
        {
            if (custom_resolution)
            {
                custom_resolution_size = glm::ivec2(screen_width, screen_height);
            }
            else
            {
                framebuffer->resize(screen_width, screen_height);
            }
        }
        if (custom_resolution)
        {
            if (ImGui::InputInt2("Resolution", &custom_resolution_size.x))
            {
                framebuffer->resize(custom_resolution_size.x, custom_resolution_size.y);
            }
        }
    }
    ImGui::End();

    framerate_layer();

    // ImGui::ShowDemoWindow();
}

void App::framerate_layer()
{
    if (framerate_history.framerate.size() > 2 * MAX_FRAMERATE_HISTORY)
    {
        framerate_history.framerate.erase(framerate_history.framerate.begin(), framerate_history.framerate.begin() + MAX_FRAMERATE_HISTORY);
        framerate_history.delta_time.erase(framerate_history.delta_time.begin(), framerate_history.delta_time.begin() + MAX_FRAMERATE_HISTORY);
        framerate_history.timestamp.erase(framerate_history.timestamp.begin(), framerate_history.timestamp.begin() + MAX_FRAMERATE_HISTORY);
    }
    // Whenever this layer is called, that probably means one exact frame has passed

    float this_frame_timestamp = elapsed;
    float this_frame_fps = 1.0f / delta_time;
    float this_frame_dt = delta_time;
    framerate_history.best_framerate = glm::max(framerate_history.best_framerate, this_frame_fps);
    framerate_history.worst_framerate = glm::min(framerate_history.worst_framerate, this_frame_fps);

    framerate_history.timestamp.push_back(this_frame_timestamp);
    framerate_history.framerate.push_back(this_frame_fps);
    framerate_history.delta_time.push_back(this_frame_dt);

    framerate_sum += this_frame_fps;
    int num_frames = framerate_history.framerate.size();
    // std::cout << "Adding " << framerate_sum << ", num frames: " << num_frames << std::endl;
    if (framerate_history.framerate.size() > MAX_FRAMERATE_HISTORY)
    {
        framerate_sum -= framerate_history.framerate[framerate_history.framerate.size() - MAX_FRAMERATE_HISTORY - 1];
        num_frames = MAX_FRAMERATE_HISTORY;
        // std::cout << "Removing first frame. Num frames: " << num_frames << std::endl;
    }

    std::memcpy(framerate_history.history_ys,
                &framerate_history.framerate[framerate_history.framerate.size() - num_frames],
                sizeof(float) * num_frames);

    ImGui::SetNextWindowPos({ 0, 380 }, ImGuiCond_FirstUseEver);
    ImGui::SetNextWindowSize({ 250, (float) screen_height - 380.0f }, ImGuiCond_FirstUseEver);
    if (ImGui::Begin("Framerate"))
    {
        ImGui::Text("Framerate: %f", framerate_history.framerate.back());

        if (framerate_history.framerate.size() > 0)
        {
            ImGui::Text("Average: %f", framerate_sum / num_frames);
            ImGui::Text("Best FPS: %f, worst FPS: %f", framerate_history.best_framerate, framerate_history.worst_framerate);
            sprintf_s(framerate_history.stress_test_desc, "Stress test %s", framerate_history.stress_test ? "ON" : "OFF");
            if (ImGui::Button(framerate_history.stress_test_desc))
            {
                framerate_history.stress_test = !framerate_history.stress_test;
            }
        }

        if (ImPlot::BeginPlot("FPS Plot"))
        {
            ImPlot::SetupAxes("record", "FPS");
            ImPlot::PlotLine("Framerate", framerate_history.history_xs, framerate_history.history_ys, num_frames);
            ImPlot::EndPlot();
        }
    }
    ImGui::End();

    // ImPlot::ShowDemoWindow();
}

void App::set_user_interface_mode(bool new_ui_mode)
{
    user_interface_mode = new_ui_mode;

    if (new_ui_mode)
    {
        // Stop capturing cursor and stop camera motion
        glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_NORMAL);
    }
    else
    {
        glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
    }
}

void App::favorite_camera_pose() const
{
    std::ofstream dumper("camera_pose.txt");
    if (!dumper.good())
    {
        std::cerr << "Bad writer for camera_pose.txt?" << std::endl;
        return;
    }

    constexpr auto max_precision = std::numeric_limits<long double>::digits10 + 1;
    dumper << std::setprecision(max_precision);
    dumper << camera.eye.x << " " << camera.eye.y << " " << camera.eye.z << std::endl;
    dumper << camera.pitch << " " << camera.yaw << " " << camera.sensitivity << " " << camera.speed << std::endl;
    dumper << camera.z_near << " " << camera.z_far << std::endl;
    dumper.close();
}

void App::restore_camera_pose()
{
    std::ifstream reader("camera_pose.txt");
    if (!reader.good())
    {
        std::cerr << "Cannot read camera_pose.txt?" << std::endl;
        return;
    }

    reader >> camera.eye.x >> camera.eye.y >> camera.eye.z
        >> camera.pitch >> camera.yaw >> camera.sensitivity >> camera.speed
        >> camera.z_near >> camera.z_far;
    reader.close();
    camera.update_components(screen_width, screen_height);    
}

void App::debug_vf() const
{
    std::cout << "Entering vector field debug mode." << std::endl;
    std::cout << "Input coordinates to sample texture." << std::endl;

    float x, y, z;
    
    while (true)
    {
        std::cin >> x >> y >> z;
        if (x < 0 || y < 0 || z < 0)
        {
            break;
        }

        float4 result = launch_sample_single_texture_3d_kernel(res.vf_tex.texture, x, y, z);
        std::cout << result.x << ", " << result.y << ", " << result.z << ", " << result.w << std::endl;
    }
}

bool App::screenshot() const
{
    namespace fs = std::filesystem;

    fs::path figs("figs");
    const auto type = fs::status(figs).type();
    if (type == fs::file_type::not_found)
    {
        if (!fs::create_directory(figs))
        {
            std::cerr << "Cannot create directory " << figs << "?" << std::endl;
            return false;
        }
    }
    if (type != fs::file_type::not_found && type != fs::file_type::directory)
    {
        std::cerr << "Figs already exists and is not folder?" << std::endl;
        return false;
    }

    const auto now = std::chrono::system_clock::now();
    time_t now_time = std::chrono::system_clock::to_time_t(now);
    tm *time_tm = localtime(&now_time);
    char time_str[256] = { 0 };
    strftime(time_str, sizeof(time_str), "%y%m%d_%H%M", time_tm);

    const auto fb_size = framebuffer->get_size();
    
    // Split '/'
    std::string vf_name = res.vf_name;
    const auto slash_pos = vf_name.find_last_of("/");
    if (slash_pos != std::string::npos)
    {
        vf_name = vf_name.substr(slash_pos + 1);
    }

    fs::path screenshot_path = figs / (vf_name + "_" + 
        (std::to_string(fb_size.x) + "x" + std::to_string(fb_size.y)) + "_" + time_str + ".jpg");

    if (!framebuffer->screenshot(screenshot_path.string()))
    {
        std::cerr << "Cannot screenshot to " << screenshot_path << "?" << std::endl;
        return false;
    }
    return true;
}
